#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>

#test branch


#define max(a,b) \  
({ __typeof__ (a) _a = (a); \  
__typeof__ (b) _b = (b); \  
_a > _b ? _a : _b; })  
  
#define min(a,b) \  
({ __typeof__ (a) _a = (a); \  
__typeof__ (b) _b = (b); \  
_a < _b ? _a : _b; })


/*
works to do
1. longitudinal movement function
 --> based on CTM function
 --> function 
    --> input : (N, maxN, vf, maxY, w{wave speed}, dt)
    --> output : y {array}
 
2. lateral movement function
 --> LC probability : logit function (~ speed diff. / vf)
 
3. vehicle transmission function
*/




int w = 15;
int dt = 5;


int maxNconst = 20;
float maxYconst = 1800;
float Vfconst = 50;
float CellLengthconst = 100;


int m = 6; //number of cells in current link
int l = 4; //maximum number of lanes in current link

/*
link newlink;
newlink.NoLane = 4;
newlink.NoCell = 6;
*/

double CFsim(newlink, w, dt)
{
	int N = newlink.N;
	int maxN = newlink.maxN;
	float Y = newlink.Y;
	float maxY = newlink.maxY;
	float V = newlink.V;
	float Vf = newlink.Vf;

	float L = newlink.CellLength;
	
	int NoCell = newlink.NoCell;
	int NoLane = newlink.NoLane;
	
	float Lmin = Vf/3.6 * dt;
	
	int l;
	int m;

	for (m = 0; m < NoCell; m++) {
		for (l = 0; l < NoLane; l++) {
			if (m == 0) {
				Y[m][l] = 1;
			} else if {
				Y[m][l] = min( min( Lmin/L[m] * N[m][l], maxY[m][l]), 
						min( maxY[m][l+1], w * dt / L * (maxN[m][l] - N[m][l] ));
			}
		
		N[m][l] += Y[m][l]
		
		}

	}



	return();
}




/*typedef struct {
	// about link
	int linkID;
	int fromNode;
	int toNode;
	int speedlimit;
	int roadlevel;	// 고속국도, 국도, 지방도
	int road_type;
	// about section
	int sectionID;
	int no_lanes;
	int cell_id;
	int speed;
	int numberOfVehicle;
	int length;
	int nextSectionID;
	int startSectionID;
	int endSectionID;
	int distanceToNode;
	int busLane;
	// yin
	int y_in;
	// lane_change
	// int lane_change;		// no need
	// insert vehicle
} lane_cell;
*/

typedef struct {
	int NoLane;
	int NoCell;
	int N[NoCell][NoLane];  		// 2D Array [NoCell	,NoLane]
	int MaxN[NoCell][NoLane];		// 2D Array [NoCell	,NoLane]
	int LC_left[NoCell][NoLane]; 	// 2D Array [NoCell	,NoLane]
	int LC_Right[NoCell][NoLane]; 	// 2D Array [NoCell	,NoLane]
	float V[NoCell][NoLane];		// 2D Array [NoCell	,NoLane]
	float Vf[NoCell][NoLane];		// 2D Array [NoCell	,NoLane]
	float Y[NoCell+1][NoLane];		// 2D Array [NoCell+1	,NoLane]
	float MaxY[NoCell+1][NoLane];		// 2D Array [NoCell	,NoLane]
	float CellLength[NoCell];
		
		
	int NextLink[NoLane]
	int NextLane[NoLane]
		
} link;


// hello this 

typedef struct {
	int nodeID;
	int type; 	// intersection, source, sink
	int speedlimit;
	int intersection_type;
	//connection_info
} node;

typedef struct {
	// current position
	int currentLane;
	int currentCell;
	int currentLink;
	int currentLinkOrder; // 
	// int distanceToNode;

	int type;
	
	int MandatoryLC;
	int moveForward;
	int moveRight;
	int moveLeft;
	
//	int freeflowspeed;
//	int minSpacing;
//	int reactionTime;
//	int driving_moment;
//	int lane_change;
	
	int path[20];  // Array of Links EX) [15, 17, 19,...,0,0] 
	int NoLinksinPath;  //size of array path path[NoLinksinPath]
	int targetLane1[]; // minimum Target Lane  EX) 2
	int targetLane2[]; // max Target Lane  Ex) 3
	
	
} vehicle;

/* 
*/


typedef struct {
	int startLinkID;
	int startSectionID;
	int startLaneID;
	int endLinkID;
	int endSectionID;
	int endLaneID;

	int greenTime;
	int LeftTime;
	int yellowTime;
	int redTime;
	int offset;
} turning_info;

__global__ void simulationStep(int loop_limit, link *linkcell, node *n,
		vehicle *v) {
	int tid = threadIdx.x;
	int i = blockIdx.x * blockDim.x + tid;

	// simulation time
		for (int current = 0; current < loop_limit; current++) {
			
			
			// Vehicle List에서 처리 
			
			
			for(int vehID = 0; vehID<size(vehList); vehID++){
				// Mandatory Lane Change 대상 차량 선정 
				veh=vehList(vehID);
				int TargetLaneLeft=vehList(vehID).targetLane1[(vehList(vehId).currentLinkOrder];
				int TargetLaneRight=vehList(vehID).targetLane2[(vehList(vehId).currentLinkOrder];
				
				if(veh.currentLane < TargetLaneLeft){veh.lanechange=1;}
				elseif(veh.currentLane < TargetLaneLeft) {veh.lanechange=-1;}
				else (veh.lanechange=0;) 	
			}
			
				// if vehicle 
				
		
				
				// Optional LC 
				
			
			
			
			
			for (int current_link = 0; current_link < sizeof(linkcell);
					i + current_link++) {
				
			//Lane Change Execution	
				
				
				
				

				// update v <= v_agent + v
				lc[current_link].speed = 60 * lc[current_link].numberOfVehicle;

				
				
				// y_out -> y_in
				lc[current_link].
					
				
				
					
					
					
				// record result

			}
										
										
			// Vehile Update 
										
			// 
			for(int vehID = 0; vehID<size(vehList); vehID++){
				vehicle_move(veh);			
			}							
										
	}
}
										
int vehicle_move(vehicle veh){
	if (veh.Lanechange = +1) {}	// Move vehicle to left lane 
	if (veh.Lanechange = -1) {} 	// Move vehicle to Right lane 	
	if (veh.moveforward = 1) {}     // Move vehicle to frent cell
}								
						

int main(void) {
	// simulation info
	int total_simulation_time = 100, period = 5;

	// set data size
	int numLaneCell = 16, gpuBlockSize = 4, lcSize = sizeof(lane_cell),
			numBytes = numLaneCell * lcSize, gpuGridSize = numLaneCell
					/ gpuBlockSize;
	int numNode = 16, nodeSize = sizeof(node), numNodesBytes = numNode
			* nodeSize;

	int numVehicle = 16, vehicleSize = sizeof(vehicle), numVehicleBytes =
			numVehicle * vehicleSize;

	int numVehicleQueue = 16, numVehicleQueueBytes = numVehicleQueue
			* vehicleSize;

	int numResult = total_simulation_time / period, numResultBytes = numResult
			* lcSize;

	// allocate memory
	lane_cell *cpuLCArray, *gpuLCArray;
	node *cpuNodeArray, *gpuNodeArray;
	vehicle *cpuVehicleArray, *gpuVehicleArray;
	lane_cell *cpuResultArray, *gpuResultArray;

	cpuLCArray = (lane_cell*) malloc(numBytes);
	cpuNodeArray = (node *) malloc(numNodesBytes);
	cpuVehicleArray = (vehicle *) malloc(numVehicleBytes);
	cpuResultArray = (lane_cell*) malloc(
			numBytes * total_simulation_time / period);

	// input initial data
	printf("%d\n", numBytes);
	printf("%d\n", numNodesBytes);
	printf("%d\n", numVehicleBytes);
	// copy host memory to device memory
	hipMalloc((void**) &gpuLCArray, numBytes);
	hipMalloc((void**) &gpuNodeArray, numNodesBytes);
	hipMalloc((void**) &gpuVehicleArray, numVehicleBytes);

	// launch kernel
	simulationStep<<<gpuGridSize, gpuBlockSize>>>(numResult, gpuLCArray,
			gpuNodeArray, gpuVehicleArray);

	// retrieve the results
	hipMemcpy(cpuResultArray, gpuResultArray, numResultBytes,
			hipMemcpyDeviceToHost);
	// 필요없음
	// hipMemcpy(cpuLCArray, gpuLCArray, numBytes, hipMemcpyDeviceToHost);
	// hipMemcpy(cpuNodeArray, gpuNodeArray, numNodesBytes, hipMemcpyDeviceToHost);
	// hipMemcpy(cpuVehicleArray, gpuNodeArray, numVehicleBytes, hipMemcpyDeviceToHost);

	// record the simulation result
	printf("simulation results:\n");
	for (int i = 0; i < numResult; ++i) {
		//printf("point.a: %f, point.b: %f\n",cpuPointArray[i].a,cpuPointArray[i].b);
	}

	printf("end");

	// deallocate memory
	free(cpuLCArray);
	free(cpuNodeArray);
	free(cpuVehicleArray);
	free(cpuResultArray);

	hipFree(gpuLCArray);
	hipFree(gpuNodeArray);
	hipFree(gpuVehicleArray);
	hipFree(gpuResultArray);

	return 0;
}
