#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>

#test branch


#define MAX(a,b) \  
({ __typeof__ (a) _a = (a); \  
__typeof__ (b) _b = (b); \  
_a > _b ? _a : _b; })  
  
#define MIN(a,b) \  
({ __typeof__ (a) _a = (a); \  
__typeof__ (b) _b = (b); \  
_a < _b ? _a : _b; })


/*
works to do
1. longitudinal movement function
 --> based on CTM function
 --> function 
    --> input : (N, maxN, vf, maxY, w{wave speed}, dt)
    --> output : y {array}
 
2. lateral movement function
 --> LC probability : logit function (~ speed diff. / vf)
 
3. vehicle transmission function
*/

int dt = 5;     //time step

int maxNconst = 20;
double maxYconst = 1800;
double Vfconst = 50;
double CellLengthconst = 100;


typedef struct {
	
	int NoLane;  	 //INPUT argument 
	int NoSection;      //INPUT argument 
	int VehMax;      //INPUT argument 
	
	int N[NoSection][NoLane];  		// 2D Array [NoCell	,NoLane]
	int MaxN[NoSection][NoLane];		// 2D Array [NoCell	,NoLane]
	int LC_Left[NoSection][NoLane]; 		// 2D Array [NoCell	,NoLane]
	int LC_Right[NoSection][NoLane]; 		// 2D Array [NoCell	,NoLane]
	double V[NoSection][NoLane];		// 2D Array [NoCell	,NoLane]
	double Y[NoSection+1][NoLane];		// 2D Array [NoCell+1	,NoLane]
	double MaxY[NoSection+1][NoLane];		// 2D Array [NoCell	,NoLane]
	double SectionLength[NoSection];
	double Vf;// Free flow speed 	
	
	// Vehicle Move 관련 
	int veh[NoSection+2][NoLane][VehMax];		// vehID per each cell (include buffer cell)
	int vehMLC[NoSection+2][NoLane][VehMax];    	// 1이면 오른쪽으로차로변경,-1이면 왼쪽으로 변경
	int vehOLC[NoSection+2][NoLane][VehMax]; 		// 1이면 오른쪽으로차로변경,-1이면 왼쪽으로 변경
	int vehMoveForward[NoSection+2][NoLane][VehMax];   //1이면 다음셀로 전진,0이면 현재셀에 머무르기
	
	// Vehicle 속성중에서 MLC 관련 속성 넣기 
	int targetLane1[NoSection+2][NoLane][VehMax]; 	// minimum Target Lane  EX) 2  타겟 레인의 하한값 설정
	int targetLane2[NoSection+2][NoLane][VehMax]; 	// max Target Lane  Ex) 3   타겟 레인 가안 값 설정 	
	
	// 시그널 넣기 
	int greenTime[NoLane];	 			// 1이면 Green signal, 0이면 Red signal
	
	// 글로벌 메모리를 잘 쓰자 -- 글로벌 메모리에서 링크 MLC 결정을 위한 차량보

	int NextConnectionSection;
	int PreviousConnectionSection;
		
} link;


typedef struct {
	
	int nodeID;
	int type; 	// intersection, source, sink
	int speedlimit;
	int intersection_type;
	//connection_info
} node;

typedef struct {
	// current position
	int currentLane;
	int currentSection;      // 현재 cell, 링크 시작점이 0,  
	int currentLink;      // 현재 링크 ID 
	int currentLinkOrder; // path[]에서 현재 링크 순서  
	// int distanceToNode;

	int type;
	
//	int MandatoryLC;
//	int moveForward;  // 1이면 시뮬레이션 시 다음셀로 차량을 이동할 필요  
	
//	int freeflowspeed;
//	int minSpacing;
//	int reactionTime;
//	int driving_moment;
	
//	int lanechange;  // 1이면 오른쪽으로 차로변경, -1이면 왼쪽으로 변경이 필요 
	
	int path[20];  // Array of Link IDs EX) [15, 17, 19,...,0,0] 
	int NoLinksinPath;  //size of array path path[NoLinksinPath]  path 의 데이터 크기 
	int targetLane1[20]; // minimum Target Lane  EX) 2  타겟 레인의 하한값 설정
	int targetLane2[20]; // max Target Lane  Ex) 3   타겟 레인 상한 값 설정 	
} vehicle;


typedef struct {
	int NoConnection;   			// 전체 커넥션 수 INPUT argument 
	int VehMax;                     	// 커넥션의 차량 수  
	int fromLinkID[NoConnection];  		// 커넥션의 fromLink ID
	int toLinkID[NoConnection];           	// 커넥션의 toLink ID
	int veh[NoConnection][VehMax];        	// 커넥션상의 차량 ID  

	int greenTime[NoConnection];	         //각 커넥션의 한 시뮬레이션 스텝 중의 신호 1: Green, 0: Red

} cennection_cell;				// 커넥션은 글로벌 메모리상에서 상주하여, 링크의 아웃풋 데이터들을 다음 링크와 공유할수 있도록 해준다. 


__global__ void simulationStep(int loop_limit, link *l, node *n,
		vehicle *v) {
	int tid = threadIdx.x;
	int i = blockIdx.x * blockDim.x + tid;

	// simulation time
	for (int current = 0; current < loop_limit; current++) {
		
		
		// read vehicle from connectCell (i'th vehicle in global memory )
		//NextCoonectionCell에서 signal 정보 읽어오기 => link
		
		
		//Previous ConnectionCell에서 Input 읽어오기 => link
		
		
		
		// 각 링크l[i]별로 Mandatory LC 처리	
		Evaluate_MLC(l[i]);	 // 
	 		
					 				
		// 각 링크l[i]별로 Optioanl LC 처리
		Evalauate_OLC(l[i]);
			
		//각 링크l[i]별로 CTM SIM 처리    
		CFsim(l[i]);

			
		// 링크별 결과 전송

		
																
		//전체 차량들에대해 셀이동 처리  
		Vehicle_Move(l[i]);	
		
		// write vehicle in connectCell
			
		
		//synchronize
		__syncthreads();								
	}
}

__device__ CFsim(link* l){
	double w = 15;  //wave speed
	
	double L = l.SectionLength;
	
	int NoSection = l.NoSection;
	int NoLane = l.NoLane;
	
	double Lmin = l.Vf/3.6 * dt;
	
	
	for (int section = 0; section < l.NoSection; section++) {
		for (int lane = 0; lane < l.NoLane; lane++) {
			l.Y[section][lane] = min( min( Lmin/L[section] * l.N[section][lane], l.maxY[section][lane]), 
					min( l.maxY[section][lane+1], w * dt / L * (l.maxN[section][lane] - l.N[section][lane] ));
		// moveforward flag update
					      
		}
	}
	
}


__device__ Evaluate_MLC(link *l){

	// --------------------------------------------------------------------------------------------------
	// Mandatory Lane Change 대상 차량 선정 및 차량 데이터베이스에 차로변경 플래그(veh.lanechange) 설정 
	// --------------------------------------------------------------------------------------------------
	for(int section = 0; section < l.NoSection; section++){
		for(int lane = 0; lane < l.NoLane; lane++){
			for (int i =0 ; i < 20; i++){
				vehicle veh=l.veh[section][lane][i]; // 차량데이터 가지고 오기 
				
				int TargetLaneLeft=veh.targetLane1[veh.currentLinkOrder];  // 타겟 레인 하한 가지고 오기 
				int TargetLaneRight=veh.targetLane2[veh.currentLinkOrder];  // 타겟 레인 상한 가지고 오기 
				
				if(veh.currentLane < TargetLaneLeft){
					veh.lanechange=1;
					// l.LC_Left[veh.currentCell][veh.currentLane]=1  
				}     // 오른쪽으로 차로 변경이 필요 
				else if(veh.currentLane > TargetLaneRight) {
					veh.lanechange=-1;
					// l[veh.currentLink].LC_Righft[veh.currentCell][veh.currentLane]=1;}  // 왼쪽으로 차로 변경이 필요
				else (veh.lanechange=0;) 
			}
		}
	}				
	// --------------------------------------------------------------------------------------------------
}

__device__ Evaluate_OLC(link* l){

	// --------------------------------------------------------------------------------------------------
	// Optional Lane Change 대상 차량 선정 및 차량 데이터베이스에 차로변경 플래그(veh.lanechange) 설정 
	// --------------------------------------------------------------------------------------------------
		
	for (int section = 0; section<l.NoCell; section++){
		for(int lane = 1; lane <l.NoLane; lane++){
			if (lane >0){
				l.LC_Left[section][lane] += (l.V[section][lane-1] - l.V[section][lane])/l.Vf;	// cell별 optional LC 확률 계산 (왼쪽)
			}
			//if (lane <l.NoLane) {
			//	l.LC_Right[section][lane] += (l.V[section][lane+1] - l.V[section][lane])/l.Vf;	// cell별 optional LC 확률 계산 (오른쪽)
			//}
			
			for (int i = 0; i <20; i++) {
				vehicle veh = l.veh[section][lane][i];  // 차량 데이터 가지고 오기
				
				srand(time(NULL));
				float prob = (rand() % 10) / 10.;	// 차량별 OLC 확률 생성
				
				if (prob <= l.LC_Left[section][lane]) {
					veh.lanechange = -1;		// 왼쪽으로 차로 변경 필요
				}
			
		}
	}
	
	
	
}
										
__device__ Vehicle_Move(link* l){
	
	for(int vehID = 0; vehID < sizeof(v); vehID++){
		vehcle veh=v[vehID]; // 차량데이터 베이스에서  가지고 오기 	
		// --------------------------------------------------------------------------------------------------
		// 차로변경이 있는 경우 차량의 현재 Cell과 링크를 업데이트 한다.
		//--------------------------------------------------------------------------------------------------
		if (veh.Lanechange = +1) {}	// Move vehicle to left lane 
		if (veh.Lanechange = -1) {} 	// Move vehicle to Right lane 	
		if (veh.moveforward = 1) {}     // Move vehicle to frent cell
	
		// --------------------------------------------------------------------------------------------------
		

		// --------------------------------------------------------------------------------------------------
		// 차량이 다음셀로 전진하는 경우 차량의 현재 Cell과 링크를 업데이트 한다.
		//--------------------------------------------------------------------------------------------------
		if (veh.moveForward==1){
			
			//글로벌 메모리의 Vehicle 정보를 업데이트 
			
			if((veh.currentCell == l[veh.currentLink].NoCell)) { 
				//connection Cell로 바꾸기
				
				// 현재 셀이 링크의 마지막셀인 경우 
				veh.currentLinkOrder++; // Path의 현재 링크 순서를 1 증가 
				veh.currentLink = veh.path[currentLinkOrder];
				veh.currentCell= 0;  // Cell position을 링크 시작점으로 \
				
				
			} else {
				// N Update  
				
				
				veh.currentCell++;   //마지막 셀이 아니면, 다음 셀로 차량을 옮긴다.
			}
		}
	}
	// --------------------------------------------------------------------------------------------------

}								
						

int main(void) {
	// simulation info
	int total_simulation_time = 100, period = 5;

	// set data size
	int numLink = 16, gpuBlockSize = 4, linkSize = sizeof(link),
			numBytes = numLink * linkSize, gpuGridSize = numLink
					/ gpuBlockSize;
	int numNode = 16, nodeSize = sizeof(node), numNodesBytes = numNode
			* nodeSize;

	int numVehicle = 16, vehicleSize = sizeof(vehicle), numVehicleBytes =
			numVehicle * vehicleSize;

	int numVehicleQueue = 16, numVehicleQueueBytes = numVehicleQueue
			* vehicleSize;

	int numResult = total_simulation_time / period, numResultBytes = numResult
			* lcSize;

	// allocate memory
	link *cpuLinkArray, *gpuLinkArray;
	node *cpuNodeArray, *gpuNodeArray;
	vehicle *cpuVehicleArray, *gpuVehicleArray;
	link *cpuResultArray, *gpuResultArray;

	cpuLinkArray = (link*) malloc(numBytes);
	cpuNodeArray = (node *) malloc(numNodesBytes);
	cpuVehicleArray = (vehicle *) malloc(numVehicleBytes);
	cpuResultArray = (link*) malloc(
			numBytes * total_simulation_time / period);

	// input initial data
	printf("%d\n", numBytes);
	printf("%d\n", numNodesBytes);
	printf("%d\n", numVehicleBytes);
	// copy host memory to device memory
	hipMalloc((void**) &gpuLinkArray, numBytes);
	hipMalloc((void**) &gpuNodeArray, numNodesBytes);
	hipMalloc((void**) &gpuVehicleArray, numVehicleBytes);

	// launch kernel
	simulationStep<<<gpuGridSize, gpuBlockSize>>>(numResult, gpuLCArray,
			gpuNodeArray, gpuVehicleArray);

	// retrieve the results
	hipMemcpy(cpuResultArray, gpuResultArray, numResultBytes,
			hipMemcpyDeviceToHost);
	// 필요없음
	// hipMemcpy(cpuLCArray, gpuLCArray, numBytes, hipMemcpyDeviceToHost);
	// hipMemcpy(cpuNodeArray, gpuNodeArray, numNodesBytes, hipMemcpyDeviceToHost);
	// hipMemcpy(cpuVehicleArray, gpuNodeArray, numVehicleBytes, hipMemcpyDeviceToHost);

	// record the simulation result
	printf("simulation results:\n");
	for (int i = 0; i < numResult; ++i) {
		//printf("point.a: %f, point.b: %f\n",cpuPointArray[i].a,cpuPointArray[i].b);
	}

	printf("end");

	// deallocate memory
	free(cpuLinkArray);
	free(cpuNodeArray);
	free(cpuVehicleArray);
	free(cpuResultArray);

	hipFree(gpuLinkArray);
	hipFree(gpuNodeArray);
	hipFree(gpuVehicleArray);
	hipFree(gpuResultArray);

	return 0;
}
