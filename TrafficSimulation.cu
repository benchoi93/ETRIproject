#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#say hello

typedef struct {
	// about link
	int linkID;
	int fromNode;
	int toNode;
	int speedlimit;
	int roadlevel;	// 고속국도, 국도, 지방도
	int road_type;

	// about section
	int sectionID;
	int no_lanes;
	int cell_id;
	int speed;
	int numberOfVehicle;
	int length;
	int nextSectionID;
	int startSectionID;
	int endSectionID;
	int distanceToNode;
	int busLane;

	// yin
	int y_in;

	// lane_change
	// int lane_change;		// no need
	// insert vehicle
} lane_cell;

typedef struct {
	int nodeID;
	int type; 	// intersection, source, sink
	int speedlimit;
	int intersection_type;
	//connection_info
} node;

typedef struct {
	// current position
	int currentLane;
	int currentSection;
	int currentLink;
	int distanceToNode;

	int type;
	int freeflowspeed;
	int minSpacing;
	int reactionTime;

	int driving_moment;
	int lane_change;
	int path[10];
} vehicle;

typedef struct {
	int startLinkID;
	int startSectionID;
	int startLaneID;
	int endLinkID;
	int endSectionID;
	int endLaneID;

	int greenTime;
	int LeftTime;
	int yellowTime;
	int redTime;
	int offset;
} turning_info;

__global__ void simulationStep(int loop_limit, lane_cell *lc, node *n,
		vehicle *v) {
	int tid = threadIdx.x;
	int i = blockIdx.x * blockDim.x + tid;

	// simulation time
		for (int current = 0; current < loop_limit; current++) {
			for (int current_lane = 0; current_lane < sizeof(lc);
					i + current_lane++) {
				// insert vehicle from vehicle queue

				// update n <= n_agent + n
				lc[current_lane].numberOfVehicle ++;

				// update v <= v_agent + v
				lc[current_lane].speed = 60 * lc[current_lane].numberOfVehicle;

				// lane change

				// y_out -> y_in
				lc[current_lane].
				// record result

			}
	}
}

int main(void) {
	// simulation info
	int total_simulation_time = 100, period = 5;

	// set data size
	int numLaneCell = 16, gpuBlockSize = 4, lcSize = sizeof(lane_cell),
			numBytes = numLaneCell * lcSize, gpuGridSize = numLaneCell
					/ gpuBlockSize;
	int numNode = 16, nodeSize = sizeof(node), numNodesBytes = numNode
			* nodeSize;

	int numVehicle = 16, vehicleSize = sizeof(vehicle), numVehicleBytes =
			numVehicle * vehicleSize;

	int numVehicleQueue = 16, numVehicleQueueBytes = numVehicleQueue
			* vehicleSize;

	int numResult = total_simulation_time / period, numResultBytes = numResult
			* lcSize;

	// allocate memory
	lane_cell *cpuLCArray, *gpuLCArray;
	node *cpuNodeArray, *gpuNodeArray;
	vehicle *cpuVehicleArray, *gpuVehicleArray;
	lane_cell *cpuResultArray, *gpuResultArray;

	cpuLCArray = (lane_cell*) malloc(numBytes);
	cpuNodeArray = (node *) malloc(numNodesBytes);
	cpuVehicleArray = (vehicle *) malloc(numVehicleBytes);
	cpuResultArray = (lane_cell*) malloc(
			numBytes * total_simulation_time / period);

	// input initial data
	printf("%d\n", numBytes);
	printf("%d\n", numNodesBytes);
	printf("%d\n", numVehicleBytes);
	// copy host memory to device memory
	hipMalloc((void**) &gpuLCArray, numBytes);
	hipMalloc((void**) &gpuNodeArray, numNodesBytes);
	hipMalloc((void**) &gpuVehicleArray, numVehicleBytes);

	// launch kernel
	simulationStep<<<gpuGridSize, gpuBlockSize>>>(numResult, gpuLCArray,
			gpuNodeArray, gpuVehicleArray);

	// retrieve the results
	hipMemcpy(cpuResultArray, gpuResultArray, numResultBytes,
			hipMemcpyDeviceToHost);
	// 필요없음
	// hipMemcpy(cpuLCArray, gpuLCArray, numBytes, hipMemcpyDeviceToHost);
	// hipMemcpy(cpuNodeArray, gpuNodeArray, numNodesBytes, hipMemcpyDeviceToHost);
	// hipMemcpy(cpuVehicleArray, gpuNodeArray, numVehicleBytes, hipMemcpyDeviceToHost);

	// record the simulation result
	printf("simulation results:\n");
	for (int i = 0; i < numResult; ++i) {
		//printf("point.a: %f, point.b: %f\n",cpuPointArray[i].a,cpuPointArray[i].b);
	}

	printf("end");

	// deallocate memory
	free(cpuLCArray);
	free(cpuNodeArray);
	free(cpuVehicleArray);
	free(cpuResultArray);

	hipFree(gpuLCArray);
	hipFree(gpuNodeArray);
	hipFree(gpuVehicleArray);
	hipFree(gpuResultArray);

	return 0;
}
